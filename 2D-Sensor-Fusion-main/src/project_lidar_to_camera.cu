#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <vector>
#include <numeric>
#include <chrono>
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda.hpp> // For OpenCV's CUDA operations, if needed later
#include <opencv2/cudaimgproc.hpp> // CUDA image processing
#include <opencv2/cudaarithm.hpp> // CUDA arithmetic operations
#include <hip/hip_runtime.h> // For CUDA runtime functions
#include <> // For CUDA kernel launch parameters

#include "structIO.hpp"

using namespace std;

__global__ void transformLidarPoints(const LidarPoint* lidarPoints, const float* RT, const float* R_rect_00, const float* P_rect_00, cv::Point* outputPoints, int numPoints) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPoints) return;

    // Load matrices into shared memory to reduce global memory access
    __shared__ float s_RT[16];
    __shared__ float s_R_rect_00[16];
    __shared__ float s_P_rect_00[12];

    if (threadIdx.x < 16) {
        s_RT[threadIdx.x] = RT[threadIdx.x];
        s_R_rect_00[threadIdx.x] = R_rect_00[threadIdx.x];
        if (threadIdx.x < 12) {
            s_P_rect_00[threadIdx.x] = P_rect_00[threadIdx.x];
        }
    }
    __syncthreads(); // Ensure all data is loaded into shared memory

    // Transform the Lidar point to homogeneous coordinates
    float X[4] = {lidarPoints[idx].x, lidarPoints[idx].y, lidarPoints[idx].z, 1.0f};

    // Conditional checks for point boundaries and reflection intensity
    float maxX = 25.0f, maxY = 6.0f, minZ = -1.4f;
    if(X[0] > maxX || X[0] < 0.0f || fabs(X[1]) > maxY || X[2] < minZ || lidarPoints[idx].r < 0.01f) {
        return; // Skip this point if it doesn't meet the criteria
    }

    // Perform matrix multiplication: RT * X using shared memory
    float Y[4] = {0};
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            Y[i] += s_RT[i * 4 + j] * X[j];
        }
    }

    // Perform matrix multiplication: R_rect_00 * Y using shared memory
    float Z[4] = {0};
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            Z[i] += s_R_rect_00[i * 4 + j] * Y[j];
        }
    }

    // Project onto image plane: P_rect_00 * Z using shared memory
    float point2D[3] = {0};
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 4; ++j) {
            point2D[i] += s_P_rect_00[i * 4 + j] * Z[j];
        }
    }

    // Convert to 2D image coordinates
    outputPoints[idx].x = static_cast<int>(point2D[0] / point2D[2]);
    outputPoints[idx].y = static_cast<int>(point2D[1] / point2D[2]);
}

void overlayGPS(cv::Mat& image, const std::string& text, const cv::Point& org) {
    int fontFace = cv::FONT_HERSHEY_SIMPLEX;
    double fontScale = 0.5;
    int thickness = 1;
    cv::Scalar color(0, 255, 0); // Green color for the text
    int lineType = cv::LINE_8;
    bool bottomLeftOrigin = false;

    // Create a transparent overlay by copying the original image
    cv::Mat overlay = image.clone();
    // Put the text on the overlay
    cv::putText(overlay, text, org, fontFace, fontScale, color, thickness, lineType, bottomLeftOrigin);

    // Blend the overlay with the original image
    double alpha = 1.0; // Transparency factor
    cv::addWeighted(overlay, alpha, image, 1 - alpha, 0, image);
}

bool readGPSData(const string& filename, float& latitude, float& longitude) {
    ifstream gpsFile(filename);
    if (!gpsFile.is_open()) {
        cerr << "Error opening GPS data file: " << filename << endl;
        return false;
    }

    string line;
    if (getline(gpsFile, line)) {
        istringstream iss(line);
        if (!(iss >> latitude >> longitude)) {
            cerr << "Error parsing GPS data line: " << line << endl;
            return false;
        }
    } else {
        cerr << "Error reading GPS data line from file: " << filename << endl;
        return false;
    }

    gpsFile.close();
    return true;
}



void convertMatToFloatArray(const cv::Mat& inputMat, float* outputArray) {
    for (int i = 0; i < inputMat.rows; ++i) {
        for (int j = 0; j < inputMat.cols; ++j) {
            outputArray[i * inputMat.cols + j] = static_cast<float>(inputMat.at<double>(i, j));
        }
    }
}

void loadCalibrationData(cv::Mat &P_rect_00, cv::Mat &R_rect_00, cv::Mat &RT)
{
    RT.at<double>(0,0) = 7.533745e-03; RT.at<double>(0,1) = -9.999714e-01; RT.at<double>(0,2) = -6.166020e-04; RT.at<double>(0,3) = -4.069766e-03;
    RT.at<double>(1,0) = 1.480249e-02; RT.at<double>(1,1) = 7.280733e-04; RT.at<double>(1,2) = -9.998902e-01; RT.at<double>(1,3) = -7.631618e-02;
    RT.at<double>(2,0) = 9.998621e-01; RT.at<double>(2,1) = 7.523790e-03; RT.at<double>(2,2) = 1.480755e-02; RT.at<double>(2,3) = -2.717806e-01;
    RT.at<double>(3,0) = 0.0; RT.at<double>(3,1) = 0.0; RT.at<double>(3,2) = 0.0; RT.at<double>(3,3) = 1.0;
    
    R_rect_00.at<double>(0,0) = 9.999239e-01; R_rect_00.at<double>(0,1) = 9.837760e-03; R_rect_00.at<double>(0,2) = -7.445048e-03; R_rect_00.at<double>(0,3) = 0.0;
    R_rect_00.at<double>(1,0) = -9.869795e-03; R_rect_00.at<double>(1,1) = 9.999421e-01; R_rect_00.at<double>(1,2) = -4.278459e-03; R_rect_00.at<double>(1,3) = 0.0;
    R_rect_00.at<double>(2,0) = 7.402527e-03; R_rect_00.at<double>(2,1) = 4.351614e-03; R_rect_00.at<double>(2,2) = 9.999631e-01; R_rect_00.at<double>(2,3) = 0.0;
    R_rect_00.at<double>(3,0) = 0; R_rect_00.at<double>(3,1) = 0; R_rect_00.at<double>(3,2) = 0; R_rect_00.at<double>(3,3) = 1;
    
    P_rect_00.at<double>(0,0) = 7.215377e+02; P_rect_00.at<double>(0,1) = 0.000000e+00; P_rect_00.at<double>(0,2) = 6.095593e+02; P_rect_00.at<double>(0,3) = 0.000000e+00;
    P_rect_00.at<double>(1,0) = 0.000000e+00; P_rect_00.at<double>(1,1) = 7.215377e+02; P_rect_00.at<double>(1,2) = 1.728540e+02; P_rect_00.at<double>(1,3) = 0.000000e+00;
    P_rect_00.at<double>(2,0) = 0.000000e+00; P_rect_00.at<double>(2,1) = 0.000000e+00; P_rect_00.at<double>(2,2) = 1.000000e+00; P_rect_00.at<double>(2,3) = 0.000000e+00;

}

bool readLidarPts(const string& filename, vector<LidarPoint>& lidarPoints) {
    std::ifstream file(filename, std::ios::binary); // Open the file in binary mode
    if (!file.is_open()) {
        cerr << "Error opening LiDAR data file: " << filename << endl;
        return false;
    }

    float buffer[4]; // Buffer to hold each point (r, x, y, z)

    while (file.read(reinterpret_cast<char*>(&buffer), sizeof(buffer))) {
        LidarPoint point;
        point.x = buffer[0]; // Reflectivity
        point.y = buffer[1]; // X coordinate
        point.z = buffer[2]; // Y coordinate
        point.r = buffer[3]; // Z coordinate

        lidarPoints.push_back(point);

        // Print out the r, x, y, and z values
        //cout << "r: " << point.r << ", x: " << point.x << ", y: " << point.y << ", z: " << point.z << endl;
    }

    file.close();
    return true;
}



void projectLidarToCamera2() {
    string basePathImages = "C:/Users/coope/Desktop/2D-Sensor-Fusion-main/images/new_images/image_00/data/";
    string basePathLidar = "C:/Users/coope/Desktop/2D-Sensor-Fusion-main/dat/new_dat/data/";
    string basePathGPS = "C:/Users/coope/Desktop/2D-Sensor-Fusion-main/dat/new_dat/oxts/data/";

    // Define a window for displaying the results
    string windowName = "Lidar Points Projected on Images";
    cv::namedWindow(windowName, cv::WINDOW_NORMAL); // Use WINDOW_AUTOSIZE for an auto-sized window

    for (int i = 0; i <= 77; i++) {
        stringstream ss;
        ss << setw(10) << setfill('0') << i;
        //stringstream sl;
        //sl << setw(4) << setfill('0') << i;
        string imageFileName = basePathImages + ss.str() + ".png";
        string lidarFileName = basePathLidar + ss.str() + ".bin";
        string gpsFileName = basePathGPS + ss.str() + ".txt";

        auto start = chrono::high_resolution_clock::now();
        cv::Mat imgHost = cv::imread(imageFileName);
        if (imgHost.empty()) {
            cerr << "Could not read image file: " << imageFileName << endl;
            continue;
        }
        cv::cuda::GpuMat imgDevice;
        imgDevice.upload(imgHost);

        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by image read/upload: " << duration.count() << " microseconds" << endl;

        // Load Lidar points from file (assuming readLidarPts is implemented correctly)
        start = chrono::high_resolution_clock::now();
        vector<LidarPoint> lidarPoints;
        if (!readLidarPts(lidarFileName, lidarPoints)) {
            cerr << "Could not read LiDAR data file: " << lidarFileName << endl;
            continue;
        }
        
        // Store Lidar points in GPU memory
        LidarPoint* d_lidarPoints;
        hipMalloc(&d_lidarPoints, lidarPoints.size() * sizeof(LidarPoint));
        hipMemcpy(d_lidarPoints, lidarPoints.data(), lidarPoints.size() * sizeof(LidarPoint), hipMemcpyHostToDevice);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by lidar data read/upload: " << duration.count() << " microseconds" << endl; //this takes awhile!

        start = chrono::high_resolution_clock::now();

        // Load and convert calibration data to GPU
        cv::Mat P_rect_00(3, 4, CV_64F), R_rect_00(4, 4, CV_64F), RT(4, 4, CV_64F);
        loadCalibrationData(P_rect_00, R_rect_00, RT);

        float h_RT[16], h_R_rect_00[16], h_P_rect_00[12];
        convertMatToFloatArray(RT, h_RT);
        convertMatToFloatArray(R_rect_00, h_R_rect_00);
        convertMatToFloatArray(P_rect_00, h_P_rect_00);

        float *d_RT, *d_R_rect_00, *d_P_rect_00;
        cv::Point *d_outputPoints;

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken to load, convert calibration data to GPU: " << duration.count() << " microseconds" << endl;

        start = chrono::high_resolution_clock::now();

        hipMalloc(&d_RT, 16 * sizeof(float));
        hipMalloc(&d_R_rect_00, 16 * sizeof(float));
        hipMalloc(&d_P_rect_00, 12 * sizeof(float));
        hipMalloc(&d_outputPoints, lidarPoints.size() * sizeof(cv::Point));

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by memory allocation: " << duration.count() << " microseconds" << endl; //end area of interest

        start = chrono::high_resolution_clock::now();

        hipMemcpy(d_RT, h_RT, 16 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_R_rect_00, h_R_rect_00, 16 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_P_rect_00, h_P_rect_00, 12 * sizeof(float), hipMemcpyHostToDevice);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by memory copy to device: " << duration.count() << " microseconds" << endl;

        start = chrono::high_resolution_clock::now();

        int threadsPerBlock = 256;
        //int blocksPerGrid = (lidarPoints.size() + threadsPerBlock - 1) / threadsPerBlock;
        int elementsPerThread = 1;
        int blocksPerGrid = (lidarPoints.size() + (threadsPerBlock * elementsPerThread) - 1) / (threadsPerBlock * elementsPerThread);

        transformLidarPoints<<<blocksPerGrid, threadsPerBlock>>>(d_lidarPoints, d_RT, d_R_rect_00, d_P_rect_00, d_outputPoints, lidarPoints.size());

        hipDeviceSynchronize();

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by kernel execution: " << duration.count() << " microseconds" << endl;

        start = chrono::high_resolution_clock::now();

        vector<cv::Point> h_outputPoints(lidarPoints.size());
        hipMemcpy(h_outputPoints.data(), d_outputPoints, lidarPoints.size() * sizeof(cv::Point), hipMemcpyDeviceToHost);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by memory copy to host: " << duration.count() << " microseconds" << endl;

        start = chrono::high_resolution_clock::now();

        cv::Mat overlayHost = imgHost.clone();
        for (const auto& pt : h_outputPoints) {
            if (pt.x >= 0 && pt.x < overlayHost.cols && pt.y >= 0 && pt.y < overlayHost.rows) {
                cv::circle(overlayHost, pt, 5, cv::Scalar(0, 255, 0), -1);
            }
        }

        float latitude, longitude;

        if (!readGPSData(gpsFileName, latitude, longitude)) {
            cerr << "Could not read GPS data for frame: " << i << endl;
            continue;
        }

         // Prepare the GPS text
        stringstream gpsText;
        gpsText << fixed << setprecision(6) << "Lat: " << latitude << ", Lon: " << longitude;

        // Overlay the GPS text on the image
        cv::Point textOrg(10, 30);
        overlayGPS(overlayHost, gpsText.str(), textOrg);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by drawing points & calling GPS data: " << duration.count() << " microseconds" << endl;

        start = chrono::high_resolution_clock::now();

        cv::cuda::GpuMat overlayDevice;
        overlayDevice.upload(overlayHost);

        cv::cuda::GpuMat resultDevice;
        cv::cuda::addWeighted(overlayDevice, 1.0, imgDevice, 0.6, 0, resultDevice);

        cv::Mat resultHost;
        resultDevice.download(resultHost);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by image processing: " << duration.count() << " microseconds" << endl;

        cv::imshow(windowName, resultHost);

        start = chrono::high_resolution_clock::now();

        hipFree(d_lidarPoints);
        hipFree(d_RT);
        hipFree(d_R_rect_00);
        hipFree(d_P_rect_00);
        hipFree(d_outputPoints);

        stop = chrono::high_resolution_clock::now();
        duration = chrono::duration_cast<chrono::microseconds>(stop - start);
        cout << "Time taken by memory deallocation: " << duration.count() << " microseconds" << endl; //start area of interest

        cv::waitKey(50);
    }
    cv::destroyAllWindows(); // Clean up window resources
}

int main()
{
    projectLidarToCamera2();
}